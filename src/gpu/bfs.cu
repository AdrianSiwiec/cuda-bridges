#include "hip/hip_runtime.h"
#include <moderngpu/context.hxx>
#include <moderngpu/kernel_intervalmove.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/memory.hxx>
using namespace mgpu;

#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <queue>
#include <vector>
using namespace std;

#include "bfs.cuh"

__global__ void UpdateDistanceAndVisitedKernel(const int* __restrict__ frontier,
                                               int frontier_size, int d,
                                               int* distance, int* visited) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < frontier_size; i += step) {
        distance[frontier[i]] = d;
        atomicOr(visited + (frontier[i] >> 5), 1 << (frontier[i] & 31));
    }
}

__global__ void CalculateFrontierStartsAndDegreesKernel(
    const int* __restrict__ nodes, const int* __restrict__ frontier, int n,
    int* node_frontier_starts, int* node_frontier_degrees) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < n; i += step) {
        node_frontier_starts[i] = nodes[frontier[i]];
        node_frontier_degrees[i] = nodes[frontier[i] + 1] - nodes[frontier[i]];
    }
}

__global__ void AdvanceFrontierPhase1Kernel(
    const int* __restrict__ edge_frontier, int edge_frontier_size,
    const int* __restrict__ visited, int* parent, int* edge_frontier_success) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < edge_frontier_size; i += step) {
        int v = edge_frontier[i];
        int success =
            (((visited[v >> 5] >> (v & 31)) & 1) == 0 && parent[v] == -1) ? 1
                                                                          : 0;
        if (success) parent[edge_frontier[i]] = i;
        edge_frontier_success[i] = success;
    }
}

__global__ void AdvanceFrontierPhase2Kernel(
    const int* __restrict__ edge_frontier, int edge_frontier_size,
    const int* __restrict__ parent, int* edge_frontier_success) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = from; i < edge_frontier_size; i += step)
        if (edge_frontier_success[i] && parent[edge_frontier[i]] != i)
            edge_frontier_success[i] = 0;
}

void getMemInfo() {
    size_t fr, tot;
    hipMemGetInfo(&fr, &tot);
    cout << fr / 1e6 << " / " << tot / 1e6 << endl;
}

void ParallelBFS(int n, int m, mem_t<int>& nodes, mem_t<int>& edges, int source,
                 mem_t<int>& distance, context_t& context) {
    mem_t<int> visited = mgpu::fill<int>(0, (n + 31) / 32, context);
    mem_t<int> parent = mgpu::fill<int>(-1, n, context);
    mem_t<int> node_frontier(n, context);
    mem_t<int> node_frontier_starts(n, context);
    mem_t<int> node_frontier_degrees(n, context);
    mem_t<int> edge_frontier(m, context);
    mem_t<int> edge_frontier_success(m, context);

    htod(node_frontier.data(), &source, 1);

    vector<int> tmp_subarray;

    // getMemInfo();

    int node_frontier_size = 1;
    int edge_frontier_size = 0;
    for (int d = 0; node_frontier_size > 0; ++d) {
        UpdateDistanceAndVisitedKernel<<<128, 128, 0, context.stream()>>>(
            node_frontier.data(), node_frontier_size, d, distance.data(),
            visited.data());
        CalculateFrontierStartsAndDegreesKernel<<<128, 128, 0,
                                                  context.stream()>>>(
            nodes.data(), node_frontier.data(), node_frontier_size,
            node_frontier_starts.data(), node_frontier_degrees.data());

        // hacking a bit
        dtoh(tmp_subarray,
             node_frontier_degrees.data() + node_frontier_size - 1, 1);
        edge_frontier_size = tmp_subarray.front();

        scan<scan_type_exc>(node_frontier_degrees.data(), node_frontier_size,
                            node_frontier_degrees.data(), context);

        dtoh(tmp_subarray,
             node_frontier_degrees.data() + node_frontier_size - 1, 1);
        edge_frontier_size += tmp_subarray.front();

        interval_gather(edges.data(), edge_frontier_size,
                        node_frontier_degrees.data(), node_frontier_size,
                        node_frontier_starts.data(), edge_frontier.data(),
                        context);
        AdvanceFrontierPhase1Kernel<<<128, 128, 0, context.stream()>>>(
            edge_frontier.data(), edge_frontier_size, visited.data(),
            parent.data(), edge_frontier_success.data());
        AdvanceFrontierPhase2Kernel<<<128, 128, 0, context.stream()>>>(
            edge_frontier.data(), edge_frontier_size, parent.data(),
            edge_frontier_success.data());

        // hacking again
        dtoh(tmp_subarray,
             edge_frontier_success.data() + edge_frontier_size - 1, 1);
        node_frontier_size = tmp_subarray.front();

        scan<scan_type_exc>(edge_frontier_success.data(), edge_frontier_size,
                            edge_frontier_success.data(), context);

        dtoh(tmp_subarray,
             edge_frontier_success.data() + edge_frontier_size - 1, 1);
        node_frontier_size += tmp_subarray.front();

        interval_expand(edge_frontier.data(), node_frontier_size,
                        edge_frontier_success.data(), edge_frontier_size,
                        node_frontier.data(), context);
    }
}
